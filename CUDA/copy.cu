
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
using namespace std;

using DATA_TYPE = float;

__global__ void copy_from_device(DATA_TYPE *dst) {
    if (threadIdx.x == 0)
        *dst = 42;
}

void test_copy_from_device() {
    DATA_TYPE src;
    DATA_TYPE *dst;

    hipMalloc(&dst, sizeof(DATA_TYPE));

    copy_from_device<<<1, 32>>>(dst);

    hipMemcpy(&src, dst, sizeof(DATA_TYPE), hipMemcpyDeviceToHost);

    assert(src == 42 && "dst not copied");
}

int main() {
    test_copy_from_device();

    return 0;
}
