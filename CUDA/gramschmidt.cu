#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <string.h>
#include <unistd.h>

#include <iostream>
#include <cassert>
using namespace std;

/* Include benchmark-specific header. */
/* Default data type is double, default size is 512. */
#include "gramschmidt.h"
#include "matrix.h"

#include "host_kernel.h"

extern "C"
{
#include "utils.h"
}

#ifndef BLOCK_DIM
#define BLOCK_DIM 32
#endif


/* Array initialization. */
static void init_array(int ni, int nj, Arr2D &A, Arr2D &R, Arr2D &Q) {
    int i, j;

    for (i = 0; i < ni; i++)
        for (j = 0; j < nj; j++) {
            A[i][j] = ((DATA_TYPE)(i+1) * (j+1)) / ni;
            Q[i][j] = 0;
        }
    for (i = 0; i < nj; i++)
        for (j = 0; j < nj; j++)
            R[i][j] = 0;
}

/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int ni, int nj, Arr2D &A, Arr2D &R, Arr2D &Q) {
    int i, j;

    for (i = 0; i < ni; i++)
        for (j = 0; j < nj; j++) {
            cout << A[i][j];
            if (i % 20 == 0)
                cout << endl;
        }
    cout << endl;
    for (i = 0; i < nj; i++)
        for (j = 0; j < nj; j++) {
            cout << R[i][j];
            if (i % 20 == 0)
                cout << endl;
        }
    cout << endl;
    for (i = 0; i < ni; i++)
        for (j = 0; j < nj; j++) {
            cout << Q[i][j];
            if (i % 20 == 0)
                cout << endl;
        }
    cout << endl;
}

/**
 *  Computes the normalization of the k-st column of A using a reduction in shared memory.
 *  The kernel is launched once, with a single block of dimension BLOCK_DIM.
 *
 *  Returns the norm of A^(k) in R[k][k]
 */
__global__ void column_norm(DeviceArr2D A, DeviceArr2D R, int k) {
    __shared__ DATA_TYPE norm[BLOCK_DIM];

    //IMPROVEMENT: bring A^(k) in shmem

    int SUBCOL_DIM = floordiv(A.y, blockDim.x);

    for (int ly=threadIdx.x; ly < threadIdx.x + SUBCOL_DIM; ly++) {
        if (ly < A.y)
            norm[threadIdx.x] += A[ly][k] * A[ly][k];
        else
            norm[threadIdx.x] = 0;
    }
    __syncthreads();

    for (int b=blockDim.x / 2; b>0; b >>= 1) {  //  funnel pattern of reduction
        if (threadIdx.x < b)
            norm[threadIdx.x] += norm[threadIdx.x + b];
        __syncthreads();
    }

    if (threadIdx.x == 0)
        R[k][k] = sqrt(norm[0]);
}

__global__ void copy_to_q(DeviceArr2D A, DeviceArr2D R, DeviceArr2D Q, int k) {
    //  Q^(k) <- normalized A^(k)
    int SUBCOL = blockIdx.x * blockDim.x;
    int tid = threadIdx.x + SUBCOL;

    if (tid < A.y)
       Q[tid][k] = A[tid][k] / R[k][k];
}

/**
 *  Update R (lower triangular matrix) by multiplying Q^(k) and A_{k+1, y}
 */
__device__ void recompute(DeviceArr2D A, DeviceArr2D R, DeviceArr2D Q, int k) {
    int x = blockIdx.x * blockDim.x + threadIdx.x + k;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Mi porto in shared memory la k-esima colonna di Q, che viene usata per ogni x 
    __shared__ DATA_TYPE qk[BLOCK_DIM];

    if (y < Q.y)
        qk[threadIdx.y] = Q[y][k];

    if (x < R.x and y < A.y) {
        if (blockIdx.y == 0 && threadIdx.y == 0)
            R[k][x] = 0;
    
        R[k][x] += A[y][x] * Q[y][k];

    }

//non lo posso fare qua nonostante le griglie abbiano la stessa dimensione, perché R_kx non è
//ancora completo
        //in base al threadID, faccio gemm tra Q e A e salvo in R[k][j]
//        for (int ly = threadIdx.y; ly < blockDim.y; ly++) {
//            r_kj_partial[threadIdx.x] += Q[ly][threadIdx.x] * A[ly][threadIdx.x];
//        }
//        atomicAdd(&R[k][threadIdx.x], r_kj_partial[threadIdx.x]);
}

__global__ void update_a(DeviceArr2D A, DeviceArr2D R, DeviceArr2D Q, int k) {

    int x = blockIdx.x * blockDim.x + threadIdx.x + k;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ DATA_TYPE qk[BLOCK_DIM];    //< k-esima colonna di Q
    __shared__ DATA_TYPE r_k[BLOCK_DIM];   //< k-esima sottoriga di R per indici di colonna da k+1 a nj (A.y)

    if (x < R.x && threadIdx.y == 0)
        r_k[threadIdx.x] = R[k][x];

    if (y < Q.y && threadIdx.y == 0)
        qk[threadIdx.y] = Q[y][k];

    if (x < R.x and y < A.y) {
        A[y][x] -= qk[threadIdx.y] * r_k[threadIdx.x];

}

/**
 *  Host function for gramschmidt computation. Kernels are launched from host with VRAM resident data
 *  TODO: stream operations
 */
void cu_gramschmidt(Arr2D &A, Arr2D &R, Arr2D &Q) {
    DeviceArr2D dA(ni, nj);
    DeviceArr2D dR(nj, nj);
    DeviceArr2D dQ(ni, nj);

    hipMemcpy(dA.arr, A.arr, sizeof(DATA_TYPE) * A.x * A.y, hipMemcpyHostToDevice);
    
    for (k=0; k<A.x; k++) {
        column_norm<<<1, BLOCK_DIM>>>(A, R, k);
        copy_to_q<<<floordiv(A.y, BLOCK_DIM), BLOCK_DIM>>>(A, R, Q, k);

        // Operations on A right edge
        dim3 block(BLOCK_DIM, BLOCK_DIM);
        dim3 column_grid(floordiv(A.x-k, BLOCK_DIM), floordiv(A.y, BLOCK_DIM));
        update_with_basis<<<column_grid, block>>>(A, R, Q, k);

        update_a<<<column_grid, block>>>(A, R, Q, k);
    }
    
    hipMemcpy(Q.arr, dQ.arr, sizeof(DATA_TYPE) * Q.x * Q.y, hipMemcpyDeviceToHost);
    hipMemcpy(R.arr, dR.arr, sizeof(DATA_TYPE) * R.x * R.y, hipMemcpyDeviceToHost);
    
    dA.free();
    dR.free();
    dQ.free();
}

int main(int argc, char** argv)
{
    /* Retrieve problem size. */
    int ni = NI;
    int nj = NJ;

    Arr2D A(ni, nj);
    Arr2D R(nj, nj);
    Arr2D Q(ni, nj);

    /* Initialize array(s). */
    init_array(ni, nj, A, R, Q);

    struct {
        struct timespec start;
        struct timespec finish;
    } rt;
    double wt;

    clock_gettime(CLOCK_REALTIME, &rt.start);
    kernel_gramschmidt(ni, nj, A, R, Q);
    clock_gettime(CLOCK_REALTIME, &rt.finish);
    wt = (rt.finish.tv_sec - rt.start.tv_sec) + 1.0e-9 * (rt.finish.tv_nsec - rt.start.tv_nsec);
    printf("gramschmidt (Host) : %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * ni * nj * nj / (1.0e9 * wt)); //TODO: compute GFLOPS correctly
    
    clock_gettime(CLOCK_REALTIME, &rt.start);
    cu_gramschmidt(A, R, Q);
    clock_gettime(CLOCK_REALTIME, &rt.finish);
    wt = (rt.finish.tv_sec - rt.start.tv_sec) + 1.0e-9 * (rt.finish.tv_nsec - rt.start.tv_nsec);
    printf("gramschmidt (Device) : %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * ni * nj * nj / (1.0e9 * wt));

    return 0;
}
