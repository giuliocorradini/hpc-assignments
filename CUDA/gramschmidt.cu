#include <math.h>
#include <stdio.h>
#include <string.h>
#include <unistd.h>

#include <iostream>
#include <cassert>
using namespace std;

/* Include benchmark-specific header. */
/* Default data type is double, default size is 512. */
#include "gramschmidt.h"
#include "matrix.h"


extern "C"
{
#include "utils.h"
}

#ifndef BLOCK_DIM
#define BLOCK_DIM 32
#endif

#include "host_kernel.h"
#include "gramschmidt_kernel.cuh"

/* Array initialization. */
static void init_array(int ni, int nj, Arr2D &A, Arr2D &R, Arr2D &Q) {
    int i, j;

    for (i = 0; i < ni; i++)
        for (j = 0; j < nj; j++) {
            A[i][j] = ((DATA_TYPE)(i+1) * (j+1)) / ni;
            Q[i][j] = 0;
        }
    for (i = 0; i < nj; i++)
        for (j = 0; j < nj; j++)
            R[i][j] = 0;
}

/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int ni, int nj, Arr2D &A, Arr2D &R, Arr2D &Q) {
    int i, j;

    for (i = 0; i < ni; i++)
        for (j = 0; j < nj; j++) {
            cout << A[i][j];
            if (i % 20 == 0)
                cout << endl;
        }
    cout << endl;
    for (i = 0; i < nj; i++)
        for (j = 0; j < nj; j++) {
            cout << R[i][j];
            if (i % 20 == 0)
                cout << endl;
        }
    cout << endl;
    for (i = 0; i < ni; i++)
        for (j = 0; j < nj; j++) {
            cout << Q[i][j];
            if (i % 20 == 0)
                cout << endl;
        }
    cout << endl;
}

/**
 *  Host function for gramschmidt computation. Kernels are launched from host with VRAM resident data
 *  TODO: stream operations
 */
void cu_gramschmidt(Arr2D &A, Arr2D &R, Arr2D &Q) {
    DeviceArr2D dA(A.x, A.y);
    DeviceArr2D dR(R.x, R.y);
    DeviceArr2D dQ(Q.x, Q.y);

    hipMemcpy(dA.arr, A.arr, sizeof(DATA_TYPE) * A.x * A.y, hipMemcpyHostToDevice);
    
    for (int k=0; k<A.x; k++) {
        column_norm<<<1, BLOCK_DIM>>>(dA, dR, k);
        copy_to_q<<<floordiv(A.y, BLOCK_DIM), BLOCK_DIM>>>(dA, dR, dQ, k);

        // Operations on A right edge
        dim3 block(BLOCK_DIM, BLOCK_DIM);
        dim3 column_grid(floordiv(A.x-k, BLOCK_DIM), floordiv(A.y, BLOCK_DIM));
        recompute<<<column_grid, block>>>(dA, dR, dQ, k);

        update_a<<<column_grid, block>>>(dA, dR, dQ, k);
    }
    
    hipMemcpy(A.arr, dA.arr, sizeof(DATA_TYPE) * A.x * A.y, hipMemcpyDeviceToHost);
    hipMemcpy(Q.arr, dQ.arr, sizeof(DATA_TYPE) * Q.x * Q.y, hipMemcpyDeviceToHost);
    hipMemcpy(R.arr, dR.arr, sizeof(DATA_TYPE) * R.x * R.y, hipMemcpyDeviceToHost);
    
    dA.free();
    dR.free();
    dQ.free();
}

int main(int argc, char** argv)
{
    /* Retrieve problem size. */
    int ni = NI;
    int nj = NJ;

    Arr2D A(ni, nj);
    Arr2D R(nj, nj);
    Arr2D Q(ni, nj);

    Arr2D Agpu(ni, nj);
    Arr2D Rgpu(nj, nj);
    Arr2D Qgpu(ni, nj);

    /* Initialize array(s). */
    init_array(ni, nj, A, R, Q);
    init_array(ni, nj, Agpu, Rgpu, Qgpu);

    struct {
        struct timespec start;
        struct timespec finish;
    } rt;
    double wt;

    clock_gettime(CLOCK_REALTIME, &rt.start);
    kernel_gramschmidt(ni, nj, A, R, Q);
    clock_gettime(CLOCK_REALTIME, &rt.finish);
    wt = (rt.finish.tv_sec - rt.start.tv_sec) + 1.0e-9 * (rt.finish.tv_nsec - rt.start.tv_nsec);
    printf("gramschmidt (Host) : %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * ni * nj * nj / (1.0e9 * wt)); //TODO: compute GFLOPS correctly
    
    clock_gettime(CLOCK_REALTIME, &rt.start);
    cu_gramschmidt(Agpu, Rgpu, Qgpu);
    clock_gettime(CLOCK_REALTIME, &rt.finish);
    wt = (rt.finish.tv_sec - rt.start.tv_sec) + 1.0e-9 * (rt.finish.tv_nsec - rt.start.tv_nsec);
    printf("gramschmidt (Device) : %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * ni * nj * nj / (1.0e9 * wt));

    for (int i=0; i<Agpu.x; i++) {
        for (int j=0; j<Agpu.y; j++)
//            cout << Agpu[i][j] << " ";
//        cout << endl;
            if (Agpu[i][j] != A[i][j]) {
                cout << "at " << i << " " << j << endl;
                cout << "gpu: " << Agpu[i][j] << " host: " << A[i][j] << endl;
            }
    }

    for (int i=0; i<Rgpu.x; i++)
        for (int j=0; j<Agpu.y; j++)
            assert(Rgpu[i][j] == R[i][j]);
    
    for (int i=0; i<Qgpu.x; i++)
        for (int j=0; j<Agpu.y; j++)
            assert(Qgpu[i][j] == Q[i][j]);
    return 0;
}
