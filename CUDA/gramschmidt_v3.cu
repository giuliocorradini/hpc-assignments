#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#include <iostream>

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif
#define NTHREADS 4

using namespace std;

/* Include benchmark-specific header. */
/* Default data type is double, default size is 512. */
#include "gramschmidt.h"
#include "matrix.h"

extern "C"
{
#include "utils.h"
}
/* Array initialization. */
static void init_array(int ni, int nj, Arr2D &A, Arr2D &R, Arr2D &Q) {
    int i, j;

    for (i = 0; i < ni; i++)
        for (j = 0; j < nj; j++) {
            A[i][j] = ((DATA_TYPE)i * j) / ni;
            Q[i][j] = ((DATA_TYPE)i * (j + 1)) / nj;
        }
    for (i = 0; i < nj; i++)
        for (j = 0; j < nj; j++)
            R[i][j] = 0.0f;//((DATA_TYPE)i * (j + 2)) / nj;
}

/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int ni, int nj, Arr2D &A, Arr2D &R, Arr2D &Q) {
    int i, j;

    for (i = 0; i < ni; i++)
        for (j = 0; j < nj; j++) {
            cout << A[i][j];
            if (i % 20 == 0)
                cout << endl;
        }
    cout << endl;
    for (i = 0; i < nj; i++)
        for (j = 0; j < nj; j++) {
            cout << R[i][j];
            if (i % 20 == 0)
                cout << endl;
        }
    cout << endl;
    for (i = 0; i < ni; i++)
        for (j = 0; j < nj; j++) {
            cout << Q[i][j];
            if (i % 20 == 0)
                cout << endl;
        }
    cout << endl;
}

/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static void kernel_gramschmidt(int ni, int nj, Arr2D &A, Arr2D &R, Arr2D &Q) {
    int i, j, k;

    DATA_TYPE nrm;

    for (k = 0; k < nj; k++) {
        // Consideriamo la colonna k-esima di A
        nrm = 0;

        //  Calcoliamo la norma di A^(k)
        for (i = 0; i < ni; i++)
            nrm += A[i][k] * A[i][k];

        //  che viene salvata in nel k-esimo elemento diagonale di R
        R[k][k] = sqrt(nrm);

        // la k-esima colonna di Q è la normalizzazione della k-esima colonna di A
        // R[k][k] è una very busy expression
        for (i = 0; i < ni; i++)
            Q[i][k] = A[i][k] / R[k][k];

        // Per ogni colonna successiva alla k-esima (definita nell'outer loop)
        for (j = k + 1; j < nj; j++) {
            R[k][j] = 0;

            // R alla riga k, colonna j è il prodotto della k-esima colonna di Q per la j-esima colonna di A
            for (i = 0; i < ni; i++)
                R[k][j] += Q[i][k] * A[i][j];

            // aggiorno la colonna i-esima di A con il prodotto element-wise tra colonna k-esima di Q e j-esima di R
            for (i = 0; i < ni; i++)
                A[i][j] = A[i][j] - Q[i][k] * R[k][j];
        }
    }
}

/**********************************************

CUDA IMPLEMENTATION

**********************************************/
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

__global__ void norma_a(DATA_TYPE *__restrict__ a, DATA_TYPE *__restrict__ r, int ni, int nj, int k) {

    //porto in memlria 32 valori di a
    __shared__ DATA_TYPE s_a_col_k[BLOCK_SIZE];

    int a_row = blockDim.y* blockIdx.y + threadIdx.y;
    int y_thread = threadIdx.y;
    int x_thread = threadIdx.x;
    if(a_row < ni){
        s_a_col_k[threadIdx.y] = a[a_row + k];
        s_a_col_k[threadIdx.y] *= s_a_col_k[threadIdx.y];
    }
    __syncthreads();

    //RIDUCZIONE AD IMBUTO
    //faccio una riduzione da 32 valori ad 1,
    //svolta prima da 16 thread, poi da 8, poi 4 poi 2 e poi 1
    //sono secessari log(32) = 5 punti di sincronizzazzione, decisamente meno rispetto
    //ad un normale calcolo di norm
    //infine viene eseguita un unica atomic add sul valore in memoria globale 
    if(a_row < ni){
        for(int i = 2; i <= blockIdx.y; i*=2){
            if(y_thread % i == 0){
                s_a_col_k[y_thread] += s_a_col_k[y_thread+(i/2)];
            }
           __syncthreads(); 
        }
        //alla fine 1 thread (il primo) svolge l'atomic add per r
        //dovrebbe bastare solo il controllo sulle x, ma viva la paranoia
        if(y_thread==0 && x_thread == 0){
            atomicAdd(&r[k*ni+k], s_a_col_k[y_thread]);
        }
    }
}

__global__ void init_col_k_q(DATA_TYPE *__restrict__ a, DATA_TYPE *__restrict__ r, DATA_TYPE *__restrict__ q, int ni, int nj, int k) {

    int a_row = blockDim.y*blockIdx.y + threadIdx.y;
    if(a_row < ni){
        q[a_row*nj + k] = a[a_row*nj + k] / sqrt(r[k*ni+k]);
    }
}
__global__ void dot_product_a_q(DATA_TYPE *__restrict__ a, DATA_TYPE *__restrict__ r, DATA_TYPE *__restrict__ q, int ni, int nj, int k) {

    
    //porto in memlria 32 valori di a
    __shared__ DATA_TYPE s_q_col_k[BLOCK_SIZE];

    //Porto in memoria condivisa le sezioni di interesse di a e q
    int a_row = blockDim.y* blockIdx.y + threadIdx.y;
    int a_col = blockDim.x* blockIdx.y + threadIdx.x;
    //coordinate del thread, per chiarezza
    int y_thread = threadIdx.y;
    int x_thread = threadIdx.x;
    if(a_row < ni){
        //DATO CHE A è letta solo una volta, non serve portarla in memoria condivisa
        s_q_col_k[threadIdx.y] = a[a_row + a_col] * q[a_row + k];
    }
    __syncthreads();

    //RIDUCZIONE AD IMBUTO
    if(a_row < ni){
        for(int i = 2; i <= blockIdx.y; i*=2){
            if(y_thread % i == 0){
                s_q_col_k[y_thread] += s_q_col_k[y_thread+(i/2)];
            }
           __syncthreads(); 
        }
        //alla fine l'ultimo del blocco thread aggiorna la memoria condivisa
        if(y_thread==0 && x_thread == 0){
            atomicAdd(&r[k*ni+a_col], s_q_col_k[y_thread]);
        }
    }
}

__global__ void update_a(DATA_TYPE *__restrict__ a, DATA_TYPE *__restrict__ r, DATA_TYPE *__restrict__ q, int ni, int nj, int k) {
    
    int a_row = blockDim.y*blockIdx.y + threadIdx.y;
    //offset dovuto a k per tenere conto del restringimento della grid
    int a_col = (k/blockDim.x) + blockDim.x*blockIdx.x + threadIdx.x;

    if(a_col > k && a_row < ni){
        //è per chiarezza, il compilatore poi propaga il valore
        DATA_TYPE result = a[a_row*ni + a_col] - q[a_row*ni + k] * r[k*ni+a_col];
        //aggiorno il valore, non c'è concorrenza stavolta
        a[a_row*ni + a_col] = result;
    }

}

int main(int argc, char** argv)
{
    /* Retrieve problem size. */
    int ni = NI;
    int nj = NJ;

    Arr2D A(ni, nj);
    Arr2D R(nj, nj);
    Arr2D Q(ni, nj);

    /* Initialize array(s). */
    init_array(ni, nj, A, R, Q);

    struct timespec rt[2];
    double wt;

    DATA_TYPE *a =A.arr;
    DATA_TYPE *q =Q.arr; 
    DATA_TYPE *r =R.arr; 

    //allocazione memoria A,R,Q (R probabilmente non serve, si può rispatmiare spazio)
    hipHostMalloc((void **)&a, sizeof(DATA_TYPE) * ni * nj);
    hipHostMalloc((void **)&r, sizeof(DATA_TYPE) * nj * nj);
    hipHostMalloc((void **)&q, sizeof(DATA_TYPE) * ni * nj);

    //allocazione memoria GPU
    DATA_TYPE *d_a, *d_r, *d_q;
    gpuErrchk(hipMalloc((void **)&d_a, sizeof(DATA_TYPE) * ni * nj));
    gpuErrchk(hipMalloc((void **)&d_r, sizeof(DATA_TYPE) * nj * nj));
    gpuErrchk(hipMalloc((void **)&d_q, sizeof(DATA_TYPE) * ni * nj));

    //READY, STEADY, RUN!!!
    clock_gettime(CLOCK_REALTIME, rt + 0);

    gpuErrchk(hipMemcpy(d_a, a, sizeof(DATA_TYPE) * ni * nj, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_r, r, sizeof(DATA_TYPE) * ni * nj, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_q, q, sizeof(DATA_TYPE) * ni * nj, hipMemcpyHostToDevice));
    //compute the factorization

    int num_blocks;
    for (int k = 0; k < nj; k++) {
        //KERNEL PER CALCOLO DI NORM A - DIM BLOCK limitata a 32*1
        //uso un thread per riga
        num_blocks = (ni + BLOCK_SIZE - 1) / BLOCK_SIZE;
        norma_a<<<num_blocks, BLOCK_SIZE>>>(d_a, d_r, ni, nj, k);
        gpuErrchk(hipPeekAtLastError());
        
        //INIZIALIZZO COLONNA k DI Q - La grid ha la stessa dimensione
        init_col_k_q<<<num_blocks, BLOCK_SIZE>>>(d_a, d_r, d_q, ni, nj, k);
        gpuErrchk(hipPeekAtLastError());
        
        //DOPO che tutti i tread hanno scritto su A setto la radice
        //serve un thread per colonna
        num_blocks = (nj + BLOCK_SIZE - 1) / BLOCK_SIZE;
        dot_product_a_q<<<num_blocks, BLOCK_SIZE>>>(d_a, d_r, d_q, ni, nj, k);
        gpuErrchk(hipPeekAtLastError());

        //AVENDO IN R IL PRODOTTO SCALARE POSSO AGGIORNARE A, stavolta con un kernel parallelo
        //le dimensioni sono le stesse dell'operazione precedente
        //la griglia si restringe con l'aumentare di k per creare solo i thread necessari
        dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
        dim3 dimGrid((nj + BLOCK_SIZE - 1 - k)/BLOCK_SIZE, ((ni + BLOCK_SIZE - 1)/BLOCK_SIZE));
        update_a<<<dimGrid, dimBlock>>>(d_a, d_r, d_q, ni, nj, k);
        gpuErrchk(hipPeekAtLastError());
    
    }

    //MEMORY BACK TO HOST
    gpuErrchk(hipMemcpy(a, d_a, sizeof(DATA_TYPE) * nj * nj, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(r, d_r, sizeof(DATA_TYPE) * nj * nj, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(q, d_q, sizeof(DATA_TYPE) * nj * nj, hipMemcpyDeviceToHost));

    
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("gramschmidt  (GPU) : %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * ni * nj * nj / (1.0e9 * wt));

   
    
    #ifdef PRINT_DEBUG
    //ritraspongo le matrici in caso si voglia stamparle
    print_array(ni, nj, A, R, Q);
    #endif

    //FREE HOST MEMORY
    hipHostFree(a);
    hipHostFree(r);
    hipHostFree(q);
    //FREE GPU MEMORY
    hipFree(d_a);
    hipFree(d_r);
    hipFree(d_q);

    return 0;
}
