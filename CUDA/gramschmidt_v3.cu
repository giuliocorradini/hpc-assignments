#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#include <iostream>

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif
#define NTHREADS 4

using namespace std;

/* Include benchmark-specific header. */
/* Default data type is double, default size is 512. */
#include "gramschmidt.h"
#include "matrix.h"

extern "C"
{
#include "utils.h"
}

/**Funzione per trasformare la matrice in trasposta */
static void transpose_matrix(int ni, int nj, DATA_TYPE* M, DATA_TYPE* M_T)
{
    // Transpose the matrix M, to read its columns into cache as rows
#pragma omp parallel for simd num_threads(NTHREADS) schedule(static) collapse(2)
    for (int i = 0; i < ni; i++) {
        for (int j = 0; j < nj; j++) {
            M_T[j * nj + i] = M[i * ni + j];
        }
    }
}


/* Array initialization. */
static void init_array(int ni, int nj, Arr2D &A, Arr2D &R, Arr2D &Q) {
    int i, j;

    for (i = 0; i < ni; i++)
        for (j = 0; j < nj; j++) {
            A[i][j] = ((DATA_TYPE)i * j) / ni;
            Q[i][j] = ((DATA_TYPE)i * (j + 1)) / nj;
        }
    for (i = 0; i < nj; i++)
        for (j = 0; j < nj; j++)
            R[i][j] = 0.0f;//((DATA_TYPE)i * (j + 2)) / nj;
}

/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int ni, int nj, Arr2D &A, Arr2D &R, Arr2D &Q) {
    int i, j;

    for (i = 0; i < ni; i++)
        for (j = 0; j < nj; j++) {
            cout << A[i][j];
            if (i % 20 == 0)
                cout << endl;
        }
    cout << endl;
    for (i = 0; i < nj; i++)
        for (j = 0; j < nj; j++) {
            cout << R[i][j];
            if (i % 20 == 0)
                cout << endl;
        }
    cout << endl;
    for (i = 0; i < ni; i++)
        for (j = 0; j < nj; j++) {
            cout << Q[i][j];
            if (i % 20 == 0)
                cout << endl;
        }
    cout << endl;
}

/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static void kernel_gramschmidt(int ni, int nj, Arr2D &A, Arr2D &R, Arr2D &Q) {
    int i, j, k;

    DATA_TYPE nrm;

    for (k = 0; k < nj; k++) {
        // Consideriamo la colonna k-esima di A
        nrm = 0;

        //  Calcoliamo la norma di A^(k)
        for (i = 0; i < ni; i++)
            nrm += A[i][k] * A[i][k];

        //  che viene salvata in nel k-esimo elemento diagonale di R
        R[k][k] = sqrt(nrm);

        // la k-esima colonna di Q è la normalizzazione della k-esima colonna di A
        // R[k][k] è una very busy expression
        for (i = 0; i < ni; i++)
            Q[i][k] = A[i][k] / R[k][k];

        // Per ogni colonna successiva alla k-esima (definita nell'outer loop)
        for (j = k + 1; j < nj; j++) {
            R[k][j] = 0;

            // R alla riga k, colonna j è il prodotto della k-esima colonna di Q per la j-esima colonna di A
            for (i = 0; i < ni; i++)
                R[k][j] += Q[i][k] * A[i][j];

            // aggiorno la colonna i-esima di A con il prodotto element-wise tra colonna k-esima di Q e j-esima di R
            for (i = 0; i < ni; i++)
                A[i][j] = A[i][j] - Q[i][k] * R[k][j];
        }
    }
}

/**********************************************

CUDA IMPLEMENTATION

**********************************************/
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

__global__ void norma_a(DATA_TYPE *__restrict__ a, DATA_TYPE *__restrict__ r, DATA_TYPE *__restrict__ q, int ni, int nj, int k) {

    cg::thread_block cta = cg::this_thread_block();

    int a_row = blockDim.y*blockIdx.y + threadIdx.y;
    if(a_row < ni){
        DATA_TYPE value = a[a_row*nj + k];
        value *= value;
        atomicAdd(&r[k*ni+k], value);

        //sync della grid
        cg::sync(cta);
        //un thread, quando tutti gli altri hanno finito di scrivere, calcola la norma
        if(blockIdx.y==0 && blockIdx.x==0 && threadIdx.y==0 && threadIdx.x==0){
            r[k*ni+k] = sqrt(r[k*ni+k]);
        }

    }

}

__global__ void init_col_k_q(DATA_TYPE *__restrict__ a, DATA_TYPE *__restrict__ r, DATA_TYPE *__restrict__ q, int ni, int nj, int k) {

    int a_row = blockDim.y*blockIdx.y + threadIdx.y;
    if(a_row < ni){
        q[a_row*nj + k] = a[a_row*nj + k] / r[k*ni+k];
    }
}
__global__ void dot_product_a_q(DATA_TYPE *__restrict__ a, DATA_TYPE *__restrict__ r, DATA_TYPE *__restrict__ q, int ni, int nj, int k) {

    //r è inizializzzata a matrice nulla
    DATA_TYPE partial_sum = 0;
    int a_row = blockDim.y*blockIdx.y + threadIdx.y;
    int a_col = blockDim.x*blockIdx.x + threadIdx.x;

    if(a_row < ni && a_col > k){
        for(int j = k; j<nj; j++){
            partial_sum += a[a_row*nj + a_col] * q[a_row*ni + k];
        }

        r[k*ni+a_col] = partial_sum;  
    }

}

__global__ void update_a(DATA_TYPE *__restrict__ a, DATA_TYPE *__restrict__ r, DATA_TYPE *__restrict__ q, int ni, int nj, int k) {
    
    int a_row = blockDim.y*blockIdx.y + threadIdx.y;
    //offset dovuto a k per tenere conto del restringimento della grid
    int a_col = (k/blockDim.x) + blockDim.x*blockIdx.x + threadIdx.x;

    if(a_col > k && a_row < ni){
        //è per chiarezza, il compilatore poi propaga il valore
        DATA_TYPE result = a[a_row*ni + a_col] - q[a_row*ni + k] * r[k*ni+a_col];
        //aggiorno il valore, non c'è concorrenza stavolta
        a[a_row*ni + a_col] = result;
    }

}

int main(int argc, char** argv)
{
    /* Retrieve problem size. */
    int ni = NI;
    int nj = NJ;

    Arr2D A(ni, nj);
    Arr2D R(nj, nj);
    Arr2D Q(ni, nj);

    /* Initialize array(s). */
    init_array(ni, nj, A, R, Q);

    struct timespec rt[2];
    double wt;

    DATA_TYPE *a =A.arr;
    DATA_TYPE *q =Q.arr; 
    DATA_TYPE *r =R.arr; 

    //allocazione memoria A,R,Q (R probabilmente non serve, si può rispatmiare spazio)
    hipHostMalloc((void **)&a, sizeof(DATA_TYPE) * ni * nj);
    hipHostMalloc((void **)&r, sizeof(DATA_TYPE) * nj * nj);
    hipHostMalloc((void **)&q, sizeof(DATA_TYPE) * ni * nj);

    //allocazione memoria GPU
    DATA_TYPE *d_a, *d_r, *d_q;
    gpuErrchk(hipMalloc((void **)&d_a, sizeof(DATA_TYPE) * ni * nj));
    gpuErrchk(hipMalloc((void **)&d_r, sizeof(DATA_TYPE) * nj * nj));
    gpuErrchk(hipMalloc((void **)&d_q, sizeof(DATA_TYPE) * ni * nj));

    //READY, STEADY, RUN!!!
    clock_gettime(CLOCK_REALTIME, rt + 0);

    gpuErrchk(hipMemcpy(d_a, a, sizeof(DATA_TYPE) * ni * nj, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_r, r, sizeof(DATA_TYPE) * ni * nj, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_q, q, sizeof(DATA_TYPE) * ni * nj, hipMemcpyHostToDevice));
    //compute the factorization

    int num_blocks;
    for (int k = 0; k < nj; k++) {
        //KERNEL PER CALCOLO DI NORM A - DIM BLOCK limitata a 32*1
        //uso un thread per riga
        num_blocks = (ni + BLOCK_SIZE - 1) / BLOCK_SIZE;
        norma_a<<<num_blocks, BLOCK_SIZE>>>(d_a, d_r, d_q, ni, nj, k);
        gpuErrchk(hipPeekAtLastError());
        
        //INIZIALIZZO COLONNA k DI Q - La grid ha la stessa dimensione
        init_col_k_q<<<num_blocks, BLOCK_SIZE>>>(d_a, d_r, d_q, ni, nj, k);
        gpuErrchk(hipPeekAtLastError());
        
        //DOPO che tutti i tread hanno scritto su A setto la radice
        //serve un thread per colonna
        num_blocks = (nj + BLOCK_SIZE - 1) / BLOCK_SIZE;
        dot_product_a_q<<<num_blocks, BLOCK_SIZE>>>(d_a, d_r, d_q, ni, nj, k);
        gpuErrchk(hipPeekAtLastError());

        //AVENDO IN R IL PRODOTTO SCALARE POSSO AGGIORNARE A, stavolta con un kernel parallelo
        //le dimensioni sono le stesse dell'operazione precedente
        //la griglia si restringe con l'aumentare di k per creare solo i thread necessari
        dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
        dim3 dimGrid((nj + BLOCK_SIZE - 1 - k)/BLOCK_SIZE, ((ni + BLOCK_SIZE - 1)/BLOCK_SIZE));
        update_a<<<dimGrid, dimBlock>>>(d_a, d_r, d_q, ni, nj, k);
        gpuErrchk(hipPeekAtLastError());
    
    }

    //MEMORY BACK TO HOST
    gpuErrchk(hipMemcpy(a, d_a, sizeof(DATA_TYPE) * nj * nj, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(r, d_r, sizeof(DATA_TYPE) * nj * nj, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(q, d_q, sizeof(DATA_TYPE) * nj * nj, hipMemcpyDeviceToHost));

    
    clock_gettime(CLOCK_REALTIME, rt + 1);
    wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    printf("gramschmidt  (GPU) : %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * ni * nj * nj / (1.0e9 * wt));

   
    
    #ifdef PRINT_DEBUG
    //ritraspongo le matrici in caso si voglia stamparle
    print_array(ni, nj, A, R, Q);
    #endif

    //FREE HOST MEMORY
    hipHostFree(a);
    hipHostFree(r);
    hipHostFree(q);
    //FREE GPU MEMORY
    hipFree(d_a);
    hipFree(d_r);
    hipFree(d_q);

    return 0;
}
